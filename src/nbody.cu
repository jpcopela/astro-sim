#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "particle_renderer.hpp"
#include "nbody.cuh"

#define DIM 512


void CHECK_CUDA(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    }
}

void setDevice() {
    hipDeviceProp_t prop;
    int dev;

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);

    cudaGLSetGLDevice(dev);

    std::cout << "Using device " << dev << std::endl;
}

hipError_t launchKernel(unsigned int numBodies, float3* positions)
 {
    int numBlocks = 1;
    dim3 threadsPerBlock(numBodies, numBodies);

    double time = glfwGetTime();

    hipError_t cudaStatus;
    testKernel<<<numBlocks, threadsPerBlock>>>(positions, time);
    cudaStatus = hipGetLastError();

    return cudaStatus;
}

__global__ void testKernel(float3* positions, double time) {
    unsigned int id = threadIdx.x + threadIdx.y * blockDim.x;
 
    positions[id].x = threadIdx.x;
    positions[id].y = threadIdx.y;
    positions[id].z = 0.0;
}


